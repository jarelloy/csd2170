#include "hip/hip_runtime.h"
#include "Tests.h"


__global__
void CudaIntAdder(int* lhs, int* rhs, int* res)
{
  res[threadIdx.x] = lhs[threadIdx.x] + rhs[threadIdx.x];
}

void IntAdder()
{
  std::vector<int> lhs{ 1,3 };
  std::vector<int> rhs{ 3,1 };
  std::vector<int> result{ 0,0 };

  int* dLHS, * dRHS, * dRes;

  size_t size{ sizeof(int) * lhs.size() };
  hipMalloc((void**)&dLHS, size);
  hipMalloc((void**)&dRHS, size);
  hipMalloc((void**)&dRes, size);

  hipMemcpy(dLHS, lhs.data(), size, hipMemcpyHostToDevice);
  hipMemcpy(dRHS, rhs.data(), size, hipMemcpyHostToDevice);

  CudaIntAdder <<< 1, 2 >>> (dLHS, dRHS, dRes);

  hipMemcpy(result.data(), dRes, size, hipMemcpyDeviceToHost);

  hipFree(dLHS);
  hipFree(dRHS);
  hipFree(dRes);

  std::cout << result[0] << ", " << result[1] << '\n';
}
