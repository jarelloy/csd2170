#include "hip/hip_runtime.h"
/*
* Copyright 2022 Digipen.  All rights reserved.
*
* Please refer to the end user license associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms
* is strictly prohibited.
*/

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>

////////////////////////////////////////////////////////////////////

#define BLOCK_SIZE 32
typedef unsigned int uint;

__global__ void heatDistrCalc(float* in, float* out, uint nRowPoints)
{

}

///not required in A1
///Shared memory kernel function for heat distribution calculation
__global__ void heatDistrCalcShm(float* in, float* out, uint nRowPoints)
{

}

__global__ void heatDistrUpdate(float* in, float* out, uint nRowPoints)
{

}

extern "C"
void heatDistrGPU(float* d_DataIn, float* d_DataOut, uint nRowPoints, uint nIter)
{
  dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 DimGrid2(ceil((nRowPoints) / BLOCK_SIZE), ceil((nRowPoints) / BLOCK_SIZE), 1);

  for (uint k = 0; k < nIter; k++) {
    heatDistrCalc <<< DimGrid2, DimBlock >>> (d_DataIn, d_DataOut, nRowPoints);
    getLastCudaError("heatDistrCalc failed\n");
    hipDeviceSynchronize();
    heatDistrUpdate <<< DimGrid2, DimBlock >>> (d_DataOut, d_DataIn, nRowPoints);
    getLastCudaError("heatDistrUpdate failed\n");
    hipDeviceSynchronize();
  }
}
